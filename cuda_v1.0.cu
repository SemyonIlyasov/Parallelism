#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include "hipcub/hipcub.hpp"
#define N 10
__global__ void five_point_model_calc(double* U_d, double* U_d_n, int n)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < n - 1 && i > 0)
	{
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		if (j < n - 1 && j > 0)
		{
			float left = U_d[i*n + j - 1];
			float right = U_d[i*n + j + 1];
			float up = U_d[(i-1)*n + j];
			float down = U_d[(i+1)*n + j];

			U_d_n[i*n + j] = 0.25 * (left + right + up + down);
		}
	}
}

int main(void)
{

double* U = (double*)malloc(N*N*sizeof(double));
double* U_n =(double*)malloc(N*N*sizeof(double));

double* U_d;
double* U_d_n;

hipMalloc(&U_d, sizeof(double)*N*N);
hipMalloc(&U_d_n, sizeof(double)*N*N);

double delta = 10.0 / (N - 1);

for (int i = 0; i < N; i++)
{
	U[i*N] = 10 + delta * i;
	U[i] = 10 + delta * i;
	U[(N-1)*N + i] = 20 + delta * i;
	U[i*N + N - 1] = 20 + delta * i;

	U_n[i*N] = U[i*N];
	U_n[i] = U[i];
	U_n[(N-1)*N + i] = U[(N-1)*N + i];
	U_n[i*N + N - 1] = U[i*N + N - 1];
}
hipMemcpy(U_d, U, N*N*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(U_d_n, U_n, N*N*sizeof(double), hipMemcpyHostToDevice);

dim3 GRID_SIZE = dim3(ceil((N + 127)/128.), 1, 1);
dim3 BLOCK_SIZE = dim3(128, 1, 1);

five_point_model_calc<<<GRID_SIZE, BLOCK_SIZE>>>(U_d, U_d_n, N);

hipMemcpy(U_n, U_d_n, N*N*sizeof(double), hipMemcpyDeviceToHost);

for(int i = 0; i < N; i++){
	for(int j = 0; j < N; j++)
		printf("%13.10f ", U_n[i][j]);
	printf("\n");
}

free(U);
free(U_n);
hipFree(U_d);
hipFree(U_d_n);

return 0;
}
